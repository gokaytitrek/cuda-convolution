#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define N 256 //Default matrix size NxN
#define A(i,j) A[(i)*cols+(j)]  // row-major layout
#define C(i,j) C[(i)*cols+(j)]  // row-major layout

__global__ void convolution(int *A, int *C)
{
	//Filter
	int filter[3][3] = { { 1, 2, 1 }, { 2, 4, 2 }, { 1, 2, 1 } };

	//Needs for row-major layout
	int cols = N + 2;
	//int i = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int threadBlockSize = (N+2)/ blockDim.x;//The amount of processing per thread

	for (int b = threadIdx.x * threadBlockSize; b < (threadIdx.x + 1) * threadBlockSize; b++){
		
		i = b;
		
		for (int j = 0; j < N + 1; j++){//columns
			
			if (0 < i && i < N + 1 && 0 < j && j < N + 1)
			{
				int value = 0;
				value = value + A(i - 1, j - 1)	*  filter[0][0];
				value = value + A(i - 1, j)		*  filter[0][1];
				value = value + A(i - 1, j + 1)	*  filter[0][2];
				value = value + A(i, j - 1)		*  filter[1][0];
				value = value + A(i, j)			*  filter[1][1];
				value = value + A(i, j + 1)		*  filter[1][2];
				value = value + A(i + 1, j - 1)	*  filter[2][0];
				value = value + A(i + 1, j)		*  filter[2][1];
				value = value + A(i + 1, j + 1)	*  filter[2][2];
				C(i, j) = value;
			}
		}
	}

}

int main(void)
{
	//Host variables
	int A[N+2][N+2] = {};//+2 for padding matrix
	int *C;
	
	//Device variables
	int *A_d = 0, *C_d = 0;

	//Needs for row-major layout
	int cols = N + 2;

	//Calculate memory size 
	int memorySize = (N + 2) * (N + 2);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Init matrix by 0
	for (int i = 0; i < N+2; i++) {
		for (int j = 0; j < N+2; j++) {
			A[i][j] = 0;
		}
	}

	//Generate random values between 0 and 9
	srand(time(NULL));
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			A[i + 1][j + 1] = rand() % 10;
		}
	}

	C = (int *)malloc(sizeof(*C)*memorySize);

	hipMalloc((void**)&A_d, sizeof(*A_d)*memorySize);
	hipMalloc((void**)&C_d, sizeof(*C_d)*memorySize);

	//Copy from host to device
	hipMemcpy(A_d, A, sizeof(*A_d)*memorySize, hipMemcpyHostToDevice);

	hipEventRecord(start);
	convolution << <1, 512 >> >(A_d, C_d);//Block-thread
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	//Copy from device to host
	hipMemcpy(C, C_d, sizeof(*C)*memorySize, hipMemcpyDeviceToHost);

	////Print result
	//for (int i = 0; i < N + 2; i++) {
	//	for (int j = 0; j < N + 2; j++) {
	//		printf("%d ", C(i, j));
	//	}
	//	printf("\n");
	//}

	//Free memory
	hipFree(C_d);
	hipFree(A_d);
	free(C);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f", milliseconds);
	return EXIT_SUCCESS;
}

